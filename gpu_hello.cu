#include<stdio.h>
#include<hip/hip_runtime.h>

// this is the kernel
// this is the function that actually runs on the GPU
__global__ void dkernel()
{
    printf("Hello World.\n");
}

int main()
{
    // kernel launch, use 1 thread
    dkernel<<<1,1>>>();
    // synchronize CPU and GPU
    hipDeviceSynchronize();
    return 0;
}
